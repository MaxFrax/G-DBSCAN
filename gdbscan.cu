#include "hip/hip_runtime.h"
#include <stdio.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if (error != hipSuccess)                                               \
    {                                                                       \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                            \
    }                                                                       \
}

__global__ void compute_degrees(float** dataset, int d, int n, int* degrees, float squaredThreshold) {
	extern __shared__ float* coordinates;

	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int degree = 0;

	if (tid >= n)
		return;

	// 1. Load in shared memory the coordinates assigned to the current thread
    // At this stage, a nice performance boots would be assigning the unused shared memory to the L1 cache
	for (int i = 0; i < d; i++) {
		coordinates[i] = dataset[i][tid];
	}

    // 2. Compare the current thread coordinates againts all the points in device memory
    // probably memory metrics will be a nightmare.
    // Can we use shared memory somehow to optimize the accesses?
	for (int item = 0; item < n; item++) {
		float sum = 0;
		for (int dim = 0; dim < d; dim++) {
			sum += powf(coordinates[dim] - dataset[dim][item], 2);
		}

		degree += (sum > squaredThreshold);
	}

    // 3. Store the computed degree in degrees
	degrees[tid] = degree;
}

int main()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);


	int n = 100000;
	int d = 2;

	float** dimensions;
	int* degrees;

    // 1. Alloc d n-dimensional arrays in unified memory.
    // One for each data dimension    
	CHECK(hipMallocManaged(&dimensions, d * sizeof(float*)));
	CHECK(hipMallocManaged(&degrees, n * sizeof(int)));

	for (int i = 0; i < d; i++) {
		float* j;
		CHECK(hipMallocManaged(&j, n * sizeof(float)));

        // 2. Fill the arrays with random data
		for (int k = 0; k < n; k++) {
			j[k] = rand() / float(RAND_MAX) * 24.f + 1.f;
		}
	}


    // 3. How do we compute grid and block size in a smart way?

    // 4. Invoke kernel
	compute_degrees << <(n / prop.maxThreadsPerBlock) + 1, prop.maxThreadsPerBlock, d * sizeof(float) >> > (dimensions, d, n, degrees, .3f);

	hipDeviceSynchronize();

	// 5. Free memory
	for (int i = 0; i < d; i++) {
		CHECK(hipFree(&dimensions[i]));
	}

	CHECK(hipFree(dimensions));
	CHECK(hipFree(degrees));

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
